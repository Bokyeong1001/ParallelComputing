
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void MatrixMulDevice(float *A, float *B, float *C, int dim) {
    // perform matrix multiplication on device
    for(int i=0;i<blockDim.x/gridDim.x;i++){
        //used block distribution method. get 32 elements in one thread 
        int tmpSum=0;
        for(int j=0;j<dim;j++){
            tmpSum += A[blockIdx.x*dim*blockDim.x/gridDim.x + i*dim +j] * B[threadIdx.x+j*dim];
        }
        C[blockIdx.x*dim*blockDim.x/gridDim.x + i*dim+ threadIdx.x] = tmpSum;
    }
    
}

void MatrixMulHost (float *A, float *B, float *C, int dim) {
    // perform matrix multiplication on host
    float a, b, sum;
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            sum = 0.0;
            for(int k = 0; k < dim; k++) {
                a = A[ i * dim + k ];
                b = B[ k * dim + j ];
                sum += a * b;
            }
            C[ i * dim + j ] = sum;
        }
    }
}
int main(void) {
    // I/O to load matrices A and B
    float *A, *B, *C, *d_A, *d_B, *d_C, *answer;
    int dim = 128;
    A = new float[dim*dim];
    B = new float[dim*dim];
    C = new float[dim*dim];
    d_A = new float[dim*dim];
    d_B = new float[dim*dim];
    d_C = new float[dim*dim];
    answer = new float[dim*dim];

    // initialize matrices A and B on host
    for(int i=0;i<dim*dim;i++){
       A[i] = float(i%dim);
    }
    for(int p=0;p<dim*dim;p++){
        B[p] = 1.0;
    }
 
    // define thread hierarchy
    int nblocks = 4;
    int tpb = 128;

    // allocate device memory
    size_t memSize;
    memSize = dim * dim * sizeof(float);
    hipMalloc( (void**) &d_A, memSize );
    hipMalloc( (void**) &d_B, memSize );
    hipMalloc( (void**) &d_C, memSize );

    // initialize device memory
    hipMemcpy(d_A, A, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, memSize, hipMemcpyHostToDevice);

    // launch kernel, perform matrix multiplication on device
    dim3 dimGrid(nblocks);
    dim3 dimBlock(tpb);
    MatrixMulDevice<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, dim);

    // retrieve results
    hipMemcpy(C, d_C, memSize, hipMemcpyDeviceToHost);

    // verify the CUDA kernel’s result
    MatrixMulHost(A, B, answer,dim);
    int error=0;
    for(int i=0;i<dim*dim;i++){
        if(answer[i]!=C[i]){
            error++;
        }
    }
    printf("Error : %d\n",error);
}  