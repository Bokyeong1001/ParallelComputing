
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void initArray(int *A, int *B) {
    int tid;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    B[tid] = A[blockDim.x - 1 - tid];
}
// initialize an array using GPU 
int main() {
    int *h_a; // pointer for host memory
    int *h_b; // pointer for host memory
    int *d_a; // pointer for device memory
    int *d_b; // pointer for device memory
    // define thread hierarchy
    int num_blocks = 1;
    int num_th_per_blk = 256;
    // allocate host and device memory
    size_t memSize;
    memSize = num_blocks * num_th_per_blk * sizeof(int);
    h_a = (int*) malloc(memSize);
    h_b = (int*) malloc(memSize);
    for(int i=0;i<num_th_per_blk;i++){
        h_a[i]=i;
    }

    hipMalloc( (void**) &d_a, memSize);
    hipMalloc( (void**) &d_b, memSize);
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); 
    // launch kernel
    dim3 dimGrid(num_blocks);
    dim3 dimBlock(num_th_per_blk);
    initArray<<< dimGrid, dimBlock >>>(d_a,d_b);
    // retrieve results
    hipMemcpy(h_b, d_b, memSize, hipMemcpyDeviceToHost) ; 
    int error=0;
    for(int i=0;i<num_th_per_blk;i++){
        if(h_b[i]!=h_a[num_th_per_blk-1-i]){
            error++;
        }
    }
    printf("%d\n",error);
}