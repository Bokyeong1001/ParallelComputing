
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reverseArrayMultiBlock(int *d_a, int *d_b) {
    d_b[blockDim.x - 1 - threadIdx.x] = d_a[threadIdx.x];
}
// reverse an array using GPU
int main() {
    int *h_a; // pointer for host memory
    int *answer;
    int *d_a; // pointer for device input
    int *d_b; // pointer for device output
    int dimA = 1024*1024; // size of array
    // define thread hierarchy
    int nblocks = 2;
    int tpb = 1024;
    // allocate host and device memory
    size_t memSize;
    memSize = dimA * sizeof(int);
    h_a = (int*) malloc(memSize);
    hipMalloc( (void**) &d_a, memSize );
    hipMalloc( (void**) &d_b, memSize );
    // initialize host arrays, copy to device
    for (int i = 0; i < dimA; i++) {
        h_a[i] = i;
    }
    answer = (int*) malloc(memSize);
    for(int i=0;i<dimA;i++){
        answer[i]=dimA-i;
    }
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); 
    // launch kernel
    dim3 dimGrid(nblocks);
    dim3 dimBlock(tpb);
    reverseArrayMultiBlock<<< dimGrid, dimBlock >>>(d_a, d_b);
    // retrieve results
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);
    int error=0;
    for(int i=0;i<dimA;i++){
        if(answer[i]!=h_a[i]){
            error++;
        }
    }
    printf("%d\n",error);
}