
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void initArray(int *A) {
    int tid;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    A[tid] = blockDim.x - tid;
}
// initialize an array using GPU 
int main() {
    int *h_a; // pointer for host memory
    int *answer;
    int *d_a; // pointer for device memory
    // define thread hierarchy
    int num_blocks = 1;
    int num_th_per_blk = 256;
    // allocate host and device memory
    size_t memSize;
    memSize = num_blocks * num_th_per_blk * sizeof(int);
    h_a = (int*) malloc(memSize);
    answer = (int*) malloc(memSize);
    for(int i=0;i<num_th_per_blk;i++){
        answer[i]=num_th_per_blk-i;
    }
    hipMalloc( (void**) &d_a, memSize);
    // launch kernel
    dim3 dimGrid(num_blocks);
    dim3 dimBlock(num_th_per_blk);
    initArray<<< dimGrid, dimBlock >>>(d_a);
    // retrieve results
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost) ; 
    int error=0;
    for(int i=0;i<num_th_per_blk;i++){
        if(answer[i]!=h_a[i]){
            error++;
        }
    }
    printf("%d\n",error);
}