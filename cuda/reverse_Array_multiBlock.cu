
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reverseArrayMultiBlock(int *d_a, int *d_b) {
    int tid;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i =0; i<512; i++){
        d_b[tid * 512 + i] = d_a[blockDim.x * blockDim.x - 1 - tid * 512 - i];
    }
}
// reverse an array using GPU
int main() {
    int *h_a; // pointer for host memory
    int *h_b;
    int *d_a; // pointer for device input
    int *d_b; // pointer for device output
    int dimA = 1024*1024; // size of array
    // define thread hierarchy
    int nblocks = 2;
    int tpb = 1024;
    // allocate host and device memory
    size_t memSize;
    memSize = dimA * sizeof(int);
    h_a = (int*) malloc(memSize);
    h_b = (int*) malloc(memSize);
    hipMalloc( (void**) &d_a, memSize );
    hipMalloc( (void**) &d_b, memSize );
    // initialize host arrays, copy to device
    for (int i = 0; i < dimA; i++) {
        h_a[i] = i;
    }
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); 
    // launch kernel
    dim3 dimGrid(nblocks);
    dim3 dimBlock(tpb);
    reverseArrayMultiBlock<<< dimGrid, dimBlock >>>(d_a, d_b);
    // retrieve results
    hipMemcpy(h_b, d_b, memSize, hipMemcpyDeviceToHost);
    int error=0;
    for(int i=0;i<dimA;i++){
        if(h_b[i]!=h_a[dimA-1-i]){
            error++;
        }
    }
    printf("%d\n",error);
}