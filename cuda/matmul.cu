#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void MatrixMulDevice(float *A, float *B, float *C, int dim) {
    // perform matrix multiplication on device
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < dim*dim){
        int tmpSum=0;
        for(int i=0;i<dim;i++){
            tmpSum += A[tid * dim + i] * B[i * dim + tid];
        }
        C[tid]=tmpSum;
    }
    
}
void MatrixMulHost (float *A, float *B, float *C, int dim) {
    float a, b, sum;
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            sum = 0.0;
            for(int k = 0; k < dim; k++) {
                a = A[ i * dim + k ];
                b = B[ k * dim + j ];
                sum += a * b;
            }
            C[ i * dim + j ] = sum;
        } 
    }
}
int main(void) {
    // I/O to load matrices A and B
    float *A, *B, *C, *d_A, *d_B, *d_C, *answer;
    int dim = 128;
    A = new float(dim*dim);
    B = new float(dim*dim);
    C = new float(dim*dim);
    d_A = new float(dim*dim);
    d_B = new float(dim*dim);
    d_C = new float(dim*dim);
    answer = new float(dim*dim);
    // initialize matrices A and B on host
    for(int i=0;i<dim*dim;i++){
        A[i] = i%dim;
        B[i] = 1;
    }
    // define thread hierarchy
    int nblocks = 4;
    int tpb = 128;
    // allocate device memory
    size_t memSize; 
    memSize = dim * dim * sizeof(float);
    hipMalloc( (void**) &d_A, memSize );
    hipMalloc( (void**) &d_B, memSize );
    hipMalloc( (void**) &d_C, memSize );
    // initialize device memory
    hipMemcpy(d_A, A, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, memSize, hipMemcpyHostToDevice);
    // launch kernel, perform matrix multiplication on device
    dim3 dimGrid(nblocks);
    dim3 dimBlock(tpb);
    MatrixMulDevice<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, dim);
    // retrieve results
    hipMemcpy(C, d_C, memSize, hipMemcpyDeviceToHost);
    // verify the CUDA kernel’s result
    MatrixMulHost(A, B, answer,dim)
    int error=0;
    for(int i=0;i<dim*dim;i++){
        if(answer[i]!=C[i]){
            error++;
        }
    }
    printf("%d\n",error);
}
